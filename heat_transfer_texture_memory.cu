#include "hip/hip_runtime.h"


__global__ void copy_const_kernel(float *iptr, const float *cptr)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    int offset = x + y * blockDim.x * gridDim.x;

    if (cptr[offset] != 0)
    {
        iptr[offset]= cptr[offset];
    }
}

__global__ void blend_kernel(float *outSrc, const float *inSrc)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    int offset = x + y * blockDim.x * gridDim.x;

    int left = offset - 1;
    int right = offset + 1;

    if (x == 0)
        left++;

    if (x == DIM - 1)
        right--;

    int top = offset - DIM;
    int bottom = offset + DIM;
    
    if (y == 0)
        top += DIM;

    if (y == DIM - 1)
        bottom -= DIM;

    outSrc[offset] = inSrc[offset] + SPEED * (inSrc[top] + inSrc[bottom] + inSrc[left] + inSrc[right] - inSrc[offset] * 4);
}
