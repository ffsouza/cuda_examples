
#include <hip/hip_runtime.h>
#include <stdio.h>

#define imin(a,b)(a<b?a:b)

const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void dot(float *a, float *b, float *c)
{
    __shared__ float cache[threadsPerBlock];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cache_index = threadIdx.x;

    float temp = 0;
 
    while(tid < N)
    {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    // set the cache values
    cache[cache_index] = temp;

    // synchronize threads in this block
    __syncthreads();

    // ----- reduce for sum ---------------------
    int i = blockDim.x/2;
    
    while(i != 0)
    {
        if(cache_index < i)
        {
            cache[cache_index] += cache[cache_index + i];
            __syncthreads();
        }
        i /= 2;
    }
        
    if (cache_index == 0)
    {
        c[blockIdx.x] = cache[0];
    }
}

int main(void)
{
    float *a, *b, c, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;

    //Allocate memory on the CPU
    a = (float*) malloc(N*sizeof(float));
    b = (float*) malloc(N*sizeof(float));
    partial_c = (float*) malloc(blocksPerGrid*sizeof(float));

    hipMalloc((void**)&dev_a, N*sizeof(float));
    hipMalloc((void**)&dev_b, N*sizeof(float));
    hipMalloc((void**)&dev_partial_c,blocksPerGrid*sizeof(float));

    //fill data
    for (int i=0; i<N; i++)
    {
        a[i] = i;
        b[i] = i*2;
    }

    //Copy the arrays on the GPU
    hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);

    dot<<<blocksPerGrid,threadsPerBlock>>>(dev_a, dev_b, dev_partial_c);

    hipMemcpy(partial_c, dev_partial_c, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost);


    c = 0;
    for(int i=0; i<blocksPerGrid; i++)
    {
        c += partial_c[i];
    }

    #define sum_squares(x)(x*(x+1)*(2*x+1)/6)

    printf("Does GPU value %.6g = %.6g?\n", c, 2 * sum_squares( (float)(N - 1) ) );

    // free memory on the GPU side
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_partial_c );

    // free memory on the CPU side
    free( a );
    free( b );
    free( partial_c );
}
